#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include "vector.hpp"
#include "body.hpp"
#include"IO.hpp"
#include <chrono>
using namespace std;

using Real = double;
const int dim = 2;
using Vec = Vector<Real, dim>;
using Particle = Body<Real, dim>;

// ===================== Compute Force =========================
template <typename T, int N>
__host__ __device__ Vector<T, N> computeForce(const Body<T, N>& a, const Body<T, N>& b)
{
    const T G = 10.0;
    Vector<T, N> r = b.getPosition() - a.getPosition();
    T dist = r.norm() + 1e-9;
    T F =- G * (a.getMass() * b.getMass()) / (dist * dist * dist);
    return F * r;
}

// ===================== CUDA Kernel: Compute Forces =========================
template <typename T, int N>
__global__ void compute_forces_kernel(
    const Body<T, N>* bodies,
    Vector<T, N>* forces,
    int numBodies)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numBodies) return;

    Vector<T, N> totalForce;
    for (int j = 0; j < numBodies; ++j)
    {
        if (i != j)
            totalForce += computeForce(bodies[i], bodies[j]);
    }
    forces[i] = totalForce;
}

// ===================== CUDA Kernel: Update Bodies =========================
template <typename T, int N>
__global__ void update_bodies_kernel(
    Body<T, N>* bodies,
    const Vector<T, N>* forces,
    int numBodies,
    T deltaT,
    int step) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numBodies) return;

    Body<T, N> b = bodies[i];

    Vector<T, N> acc = b.getAcceleration();
    Vector<T, N> vel = b.getVelocity();
    Vector<T, N> pos = b.getPosition();

    vel += 0.5 * acc * deltaT;
    pos += vel * deltaT;
    acc = forces[i] / b.getMass();
    vel += 0.5 * acc * deltaT;

    b.setAcceleration(acc);
    b.setVelocity(vel);
    b.setPosition(pos);

    bodies[i] = b;

    // Debug print
    
}

// ===================== Host Driver =========================
void run_simulation(std::vector<Particle>& h_bodies, Real deltaT, int steps, const std::string& output_filename)
{
    int numBodies = h_bodies.size();
    int threadsPerBlock = 256;
    int blocks = (numBodies + threadsPerBlock - 1) / threadsPerBlock;

    Particle* d_bodies;
    Vec* d_forces;

    hipMalloc(&d_bodies, sizeof(Particle) * numBodies);
    hipMemcpy(d_bodies, h_bodies.data(), sizeof(Particle) * numBodies, hipMemcpyHostToDevice);

    hipMalloc(&d_forces, sizeof(Vec) * numBodies);
    hipMemset(d_forces, 0, sizeof(Vec) * numBodies);

    std::ofstream fout(output_filename);
    fout << "step,id,x,y\n";

    std::cout << "Launching kernel with blocks=" << blocks << ", threadsPerBlock=" << threadsPerBlock << std::endl;

    for (int step = 0; step < steps; ++step)
    {
        compute_forces_kernel<Real, dim> << <blocks, threadsPerBlock >> > (d_bodies, d_forces, numBodies);
        hipError_t err1 = hipGetLastError();
        if (err1 != hipSuccess) {
            std::cerr << "compute_forces_kernel launch failed: " << hipGetErrorString(err1) << std::endl;
        }
        update_bodies_kernel<Real, dim> << <blocks, threadsPerBlock >> > (d_bodies, d_forces, numBodies, deltaT, step);
        hipError_t err2 = hipGetLastError();
        if (err2 != hipSuccess) {
            std::cerr << "update_bodies_kernel launch failed: " << hipGetErrorString(err2) << std::endl;
        }
        hipDeviceSynchronize();

        if (step % 10 == 0)
        {
            hipMemcpy(h_bodies.data(), d_bodies, sizeof(Particle) * numBodies, hipMemcpyDeviceToHost);
            write_trajectory_csv_frame<Real, dim>(fout, h_bodies, step);
        }

    }

    hipMemcpy(h_bodies.data(), d_bodies, sizeof(Particle) * numBodies, hipMemcpyDeviceToHost);

    fout.close();
    hipFree(d_bodies);
    hipFree(d_forces);
}

// ===================== Example Main =========================
int main()
{

    std::vector<Particle> bodies = read_bodies_from_txt<Real, dim>("input.txt");
    std::cout << "Read " << bodies.size() << " particles from file.\n";
    auto start = std::chrono::high_resolution_clock::now();
    run_simulation(bodies, 0.01, 10000, "trajectory.csv");
    auto end = std::chrono::high_resolution_clock::now();

    // compute timecost
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);

    std::cout << "Time duration: " << duration.count() << " ms" << std::endl;
  
    print_bodies(bodies);
    

    return 0;
}
